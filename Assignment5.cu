#include "hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define l1 long long int

const int Block_Size = 1024;

__global__ void Inclusive_Scan(l1 *d_in, l1* d_out)
{
    __shared__ l1 sh_array[Block_Size];

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;

   
    sh_array[tid] = d_in[id];

    __syncthreads();

    for(int step = 1; step <= Block_Size; step *= 2)
    {
        if(tid >= step)
        {
            l1 temp = sh_array[tid-step];
            __syncthreads();
            sh_array[tid] =max( temp,sh_array[tid]);
        }
        __syncthreads();
    }
    __syncthreads();

    d_in[id] = sh_array[tid];
    __syncthreads();

     if(tid == (Block_Size - 1))
        d_out[bid] = d_in[id];

    __syncthreads();
}


__global__ void Add(l1* d_in, l1* d_out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;

    if(bid > 0)
        d_in[id] = max(d_out[bid-1],d_in[id]);

    __syncthreads();
}

int main()
{
    l1 *h_in, *h_scan;

    int Size;
    cout << "Enter size of array\n";
    cin >> Size;

    int Reduced_Size = (int)ceil(1.0*Size/Block_Size);  
    int Array_Bytes = Size * sizeof(l1);
    int Reduced_Array_Bytes = Reduced_Size * sizeof(l1);

    h_in = (l1*)malloc(Array_Bytes);
    h_scan = (l1*)malloc(Array_Bytes);

    //Random nos
    srand(time(0));
    for(l1 i=0; i<Size; i++)
    {
        h_in[i] = rand()%10;
    }

    l1 *d_in, *d_out, *d_sum;

    hipMalloc((void**)&d_in, Reduced_Size*Block_Size*sizeof(l1));  
   
   hipMalloc((void**)&d_out, Reduced_Array_Bytes);
    hipMalloc((void**)&d_sum, sizeof(l1));

    hipMemcpy(d_in, h_in, Array_Bytes, hipMemcpyHostToDevice);

    Inclusive_Scan <<< Reduced_Size, Block_Size >>> (d_in, d_out);
   
    if(Size > Block_Size)
    {
        Inclusive_Scan <<< 1, Block_Size>>> (d_out, d_sum);
        Compare <<< Reduced_Size, Block_Size >>> (d_in, d_out);
    }

    hipMemcpy(h_scan, d_in, Array_Bytes, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

   
    l1 *pref;
    pref = (l1*)malloc(Array_Bytes);
    pref[0] = h_in[0];
    for(l1 i=1; i<Size; i++)
        pref[i] = max(pref[i-1] , h_in[i]);

    l1 flag = 0;
    for(l1 i=0; i<Size; i++)
    {
        if(h_scan[i] != pref[i])
        {
            flag = 1;
            break;
        }
    }
    if(flag == 0)
        cout << "!\n";
}
